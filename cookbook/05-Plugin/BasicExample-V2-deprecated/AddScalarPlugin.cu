#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "AddScalarPlugin.h"

// kernel for GPU
__global__ void addScalarKernel(const float *input, float *output, const float scalar, const int nElement)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= nElement)
        return;

    float _1      = input[index];
    float _2      = _1 + scalar;
    output[index] = _2;
}

namespace nvinfer1
{
// class AddScalarPlugin
AddScalarPlugin::AddScalarPlugin(float const scalar)
{
    WHERE_AM_I();
    m.scalar = scalar;
}

AddScalarPlugin::AddScalarPlugin(const void *buffer, size_t const length)
{
    WHERE_AM_I();
    memcpy(&m, buffer, sizeof(m));
}

AddScalarPlugin::~AddScalarPlugin()
{
    WHERE_AM_I();
}

const char *AddScalarPlugin::getPluginType() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_NAME;
}

const char *AddScalarPlugin::getPluginVersion() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_VERSION;
}

int32_t AddScalarPlugin::getNbOutputs() const noexcept
{
    WHERE_AM_I();
    return 1;
}

int32_t AddScalarPlugin::initialize() noexcept
{
    WHERE_AM_I();
    return 0;
}

void AddScalarPlugin::terminate() noexcept
{
    WHERE_AM_I();
    return;
}

size_t AddScalarPlugin::getSerializationSize() const noexcept
{
    WHERE_AM_I();
    return sizeof(m);
}

void AddScalarPlugin::serialize(void *buffer) const noexcept
{
    WHERE_AM_I();
    memcpy(buffer, &m, sizeof(m));
    return;
}

void AddScalarPlugin::destroy() noexcept
{
    WHERE_AM_I();
    delete this;
    return;
}

void AddScalarPlugin::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I();
    mNamespace = std::string(pluginNamespace);
    return;
}

const char *AddScalarPlugin::getPluginNamespace() const noexcept
{
    WHERE_AM_I();
    return mNamespace.c_str();
}

DataType AddScalarPlugin::getOutputDataType(int32_t index, DataType const *inputTypes, int32_t nbInputs) const noexcept
{
    WHERE_AM_I();
    return inputTypes[0];
}

void AddScalarPlugin::attachToContext(cudnnContext *contextCudnn, cublasContext *contextCublas, IGpuAllocator *gpuAllocator) noexcept
{
    WHERE_AM_I();
    return;
}

void AddScalarPlugin::detachFromContext() noexcept
{
    WHERE_AM_I();
    return;
}

IPluginV2DynamicExt *AddScalarPlugin::clone() const noexcept
{
    WHERE_AM_I();
    AddScalarPlugin *p = new AddScalarPlugin(m.scalar);
    p->setPluginNamespace(mNamespace.c_str());
    return p;
}

DimsExprs AddScalarPlugin::getOutputDimensions(int32_t outputIndex, const DimsExprs *inputs, int32_t nbInputs, IExprBuilder &exprBuilder) noexcept
{
    WHERE_AM_I();
    return inputs[0];
}

bool AddScalarPlugin::supportsFormatCombination(int32_t pos, const PluginTensorDesc *inOut, int32_t nbInputs, int32_t nbOutputs) noexcept
{
    WHERE_AM_I();
    bool res;
    switch (pos)
    {
    case 0:
        res = inOut[0].type == DataType::kFLOAT && inOut[0].format == TensorFormat::kLINEAR;
        break;
    case 1:
        res = inOut[1].type == inOut[0].type && inOut[1].format == inOut[0].format;
        break;
    default: // should NOT be here!
        res = false;
    }
#ifdef DEBUG
    // inOut is PluginTensorDesc rather than DynamicPluginTensorDesc in PluginV2
    // so we can not use PRINT_FORMAT_COMBINATION() here
    std::cout << "    pos=" << pos << ":[";
    for (int i = 0; i < nbInputs + nbOutputs; ++i)
    {
        std::cout << dataTypeToString(inOut[i].type) << ",";
    }
    std::cout << "],[";
    for (int i = 0; i < nbInputs + nbOutputs; ++i)
    {
        std::cout << formatToString(inOut[i].format) << ",";
    }
    std::cout << "]->";
    std::cout << "res=" << res << std::endl;
#endif
    return res;
}

void AddScalarPlugin::configurePlugin(const DynamicPluginTensorDesc *in, int32_t nbInputs, const DynamicPluginTensorDesc *out, int32_t nbOutputs) noexcept
{
    WHERE_AM_I();
    return;
}

size_t AddScalarPlugin::getWorkspaceSize(const PluginTensorDesc *inputs, int32_t nbInputs, const PluginTensorDesc *outputs, int32_t nbOutputs) const noexcept
{
    WHERE_AM_I();
    return 0;
}

int32_t AddScalarPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int nElement = 1;
    for (int i = 0; i < inputDesc[0].dims.nbDims; ++i)
    {
        nElement *= inputDesc[0].dims.d[i];
    }
    dim3 grid(CEIL_DIVIDE(nElement, 256), 1, 1), block(256, 1, 1);
    addScalarKernel<<<grid, block, 0, stream>>>(reinterpret_cast<const float *>(inputs[0]), reinterpret_cast<float *>(outputs[0]), m.scalar, nElement);
    return 0;
}

// class AddScalarPluginCreator
PluginFieldCollection    AddScalarPluginCreator::mFC {};
std::vector<PluginField> AddScalarPluginCreator::mPluginAttributes {};

AddScalarPluginCreator::AddScalarPluginCreator()
{
    WHERE_AM_I();
    mPluginAttributes.clear();
    mPluginAttributes.emplace_back(PluginField("scalar", nullptr, PluginFieldType::kFLOAT32, 1));
    mFC.nbFields = mPluginAttributes.size();
    mFC.fields   = mPluginAttributes.data();
}

AddScalarPluginCreator::~AddScalarPluginCreator()
{
    WHERE_AM_I();
}

IPluginV2DynamicExt *AddScalarPluginCreator::createPlugin(const char *name, const PluginFieldCollection *fc) noexcept
{
    WHERE_AM_I();
    float scalar {0.0f};
    for (int32_t i = 0; i < fc->nbFields; ++i)
    {
        auto const fieldName(fc->fields[i].name);
        if (std::strcmp(fieldName, "scalar") == 0)
        {
            scalar = *static_cast<float const *>(fc->fields[i].data);
        }
    }
    AddScalarPlugin *p = new AddScalarPlugin(scalar);
    p->setPluginNamespace(mNamespace.c_str());
    return p;
}

const char *AddScalarPluginCreator::getPluginName() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_NAME;
}

const char *AddScalarPluginCreator::getPluginVersion() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_VERSION;
}

const PluginFieldCollection *AddScalarPluginCreator::getFieldNames() noexcept
{
    WHERE_AM_I();
    return &mFC;
}

IPluginV2DynamicExt *AddScalarPluginCreator::deserializePlugin(const char *name, const void *serialData, size_t serialLength) noexcept
{
    WHERE_AM_I();
    AddScalarPlugin *pObj = new AddScalarPlugin(serialData, serialLength);
    return pObj;
}

void AddScalarPluginCreator::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I();
    mNamespace = std::string(pluginNamespace);
    return;
}

const char *AddScalarPluginCreator::getPluginNamespace() const noexcept
{
    WHERE_AM_I();
    return mNamespace.c_str();
}

REGISTER_TENSORRT_PLUGIN(AddScalarPluginCreator);

} // namespace nvinfer1
