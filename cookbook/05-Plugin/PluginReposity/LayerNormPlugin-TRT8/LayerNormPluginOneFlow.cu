#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "LayerNormPluginOneFlow.h"

// ALIGNPTR
int8_t *alignPtr(int8_t *ptr, uintptr_t to)
{
    uintptr_t addr = (uintptr_t)ptr;
    if (addr % to)
    {
        addr += to - addr % to;
    }
    return (int8_t *)addr;
}

// NEXTWORKSPACEPTR
int8_t *nextWorkspacePtr(int8_t *ptr, uintptr_t previousWorkspaceSize)
{
    uintptr_t addr = (uintptr_t)ptr;
    addr += previousWorkspaceSize;
    return alignPtr((int8_t *)addr, CUDA_MEM_ALIGN);
}

namespace nvinfer1
{
// class LayerNormPluginV5
LayerNormPluginV5::LayerNormPluginV5(const std::string &name, float epsilon):
    name_(name)
{
    WHERE_AM_I();
    m_.epsilon = epsilon;
}

LayerNormPluginV5::LayerNormPluginV5(const std::string &name, const void *buffer, size_t length):
    name_(name)
{
    WHERE_AM_I();
    memcpy(&m_, buffer, sizeof(m_));
}

LayerNormPluginV5::~LayerNormPluginV5()
{
    WHERE_AM_I();
}

IPluginV2DynamicExt *LayerNormPluginV5::clone() const noexcept
{
    WHERE_AM_I();
    auto p = new LayerNormPluginV5(name_, &m_, sizeof(m_));
    p->setPluginNamespace(namespace_.c_str());
    return p;
}

int32_t LayerNormPluginV5::getNbOutputs() const noexcept
{
    WHERE_AM_I();
    return 1;
}

DataType LayerNormPluginV5::getOutputDataType(int32_t index, DataType const *inputTypes, int32_t nbInputs) const noexcept
{
    WHERE_AM_I();
    return DataType::kFLOAT;
}

DimsExprs LayerNormPluginV5::getOutputDimensions(int32_t outputIndex, const DimsExprs *inputs, int32_t nbInputs, IExprBuilder &exprBuilder) noexcept
{
    WHERE_AM_I();
    return inputs[0];
}

bool LayerNormPluginV5::supportsFormatCombination(int32_t pos, const PluginTensorDesc *inOut, int32_t nbInputs, int32_t nbOutputs) noexcept
{
    WHERE_AM_I();
    switch (pos)
    {
    case 0:
    case 1:
        return (inOut[pos].type == DataType::kFLOAT || inOut[pos].type == DataType::kHALF) && inOut[pos].format == TensorFormat::kLINEAR;
    default: // should NOT be here!
        return false;
    }
    return false;
}

void LayerNormPluginV5::configurePlugin(const DynamicPluginTensorDesc *in, int32_t nbInputs, const DynamicPluginTensorDesc *out, int32_t nbOutputs) noexcept
{
    WHERE_AM_I();
    return;
}

size_t LayerNormPluginV5::getWorkspaceSize(const PluginTensorDesc *inputs, int32_t nbInputs, const PluginTensorDesc *outputs, int32_t nbOutputs) const noexcept
{
    WHERE_AM_I();
    int          nBlock        = inputs[0].dims.d[0] * inputs[0].dims.d[1];
    const size_t element_size  = (inputs[0].type == DataType::kFLOAT) ? sizeof(float) : sizeof(__half);
    size_t       workspaceSize = ALIGN_TO(nBlock * element_size, CUDA_MEM_ALIGN) * 2;
    //realSize = nBlock * element_size;
    //workspaceSize += ALIGN_TO(realSize, CUDA_MEM_ALIGN);
    return workspaceSize;
}

int32_t LayerNormPluginV5::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    // #rows
    int nBlock = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1];
    // #cols
    int nValuePerBlock = inputDesc[0].dims.d[inputDesc[0].dims.nbDims - 1];

    auto *    mean              = reinterpret_cast<float *>(workspace);
    uintptr_t mean_size         = ALIGN_TO(nBlock * sizeof(float), CUDA_MEM_ALIGN);
    auto *    inv_variance      = reinterpret_cast<float *>(nextWorkspacePtr(reinterpret_cast<int8_t *>(mean), mean_size));
    uintptr_t inv_variance_size = mean_size;
    if (inputDesc[0].type == DataType::kFLOAT && outputDesc[0].type == DataType::kFLOAT)
    {
        oneflow::cuda::layer_norm::DirectLoad<float, float>  load((float *)inputs[0], nValuePerBlock);
        oneflow::cuda::layer_norm::DirectStore<float, float> store((float *)outputs[0], nValuePerBlock);
        oneflow::cuda::layer_norm::DispatchLayerNorm<decltype(load), decltype(store), float>(stream, load, store, nBlock, nValuePerBlock, m_.epsilon, mean, inv_variance);
    }
    else if (inputDesc[0].type == DataType::kFLOAT && outputDesc[0].type == DataType::kHALF)
    {
        oneflow::cuda::layer_norm::DirectLoad<float, float> load((float *)inputs[0], nValuePerBlock);
        oneflow::cuda::layer_norm::DirectStore<float, half> store((half *)outputs[0], nValuePerBlock);
        oneflow::cuda::layer_norm::DispatchLayerNorm<decltype(load), decltype(store), float>(stream, load, store, nBlock, nValuePerBlock, m_.epsilon, mean, inv_variance);
    }
    else if (inputDesc[0].type == DataType::kHALF && outputDesc[0].type == DataType::kFLOAT)
    {
        oneflow::cuda::layer_norm::DirectLoad<half, float>   load((half *)inputs[0], nValuePerBlock);
        oneflow::cuda::layer_norm::DirectStore<float, float> store((float *)outputs[0], nValuePerBlock);
        oneflow::cuda::layer_norm::DispatchLayerNorm<decltype(load), decltype(store), float>(stream, load, store, nBlock, nValuePerBlock, m_.epsilon, mean, inv_variance);
    }
    else if (inputDesc[0].type == DataType::kHALF && outputDesc[0].type == DataType::kHALF)
    {
        oneflow::cuda::layer_norm::DirectLoad<half, float>  load((half *)inputs[0], nValuePerBlock);
        oneflow::cuda::layer_norm::DirectStore<float, half> store((half *)outputs[0], nValuePerBlock);
        oneflow::cuda::layer_norm::DispatchLayerNorm<decltype(load), decltype(store), float>(stream, load, store, nBlock, nValuePerBlock, m_.epsilon, mean, inv_variance);
    }
    else
    {
        printf("[LayerNormPluginV5 ERROR] Should never reach here\n");
    }
    return 0;
}

void LayerNormPluginV5::destroy() noexcept
{
    WHERE_AM_I();
    delete this;
    return;
}

int32_t LayerNormPluginV5::initialize() noexcept
{
    WHERE_AM_I();
    return 0;
}

void LayerNormPluginV5::terminate() noexcept
{
    WHERE_AM_I();
    return;
}

size_t LayerNormPluginV5::getSerializationSize() const noexcept
{
    WHERE_AM_I();
    return sizeof(m_);
}

void LayerNormPluginV5::serialize(void *buffer) const noexcept
{
    WHERE_AM_I();
    memcpy(buffer, &m_, sizeof(m_));
    return;
}

void LayerNormPluginV5::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I();
    namespace_ = pluginNamespace;
    return;
}

const char *LayerNormPluginV5::getPluginNamespace() const noexcept
{
    WHERE_AM_I();
    return namespace_.c_str();
}

const char *LayerNormPluginV5::getPluginType() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_NAME;
}

const char *LayerNormPluginV5::getPluginVersion() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_VERSION;
}

void LayerNormPluginV5::attachToContext(cudnnContext *contextCudnn, cublasContext *contextCublas, IGpuAllocator *gpuAllocator) noexcept
{
    WHERE_AM_I();
    return;
}

void LayerNormPluginV5::detachFromContext() noexcept
{
    WHERE_AM_I();
    return;
}

// class LayerNormPluginV5Creator
PluginFieldCollection    LayerNormPluginV5Creator::fc_ {};
std::vector<PluginField> LayerNormPluginV5Creator::attr_;

LayerNormPluginV5Creator::LayerNormPluginV5Creator()
{
    WHERE_AM_I();
    attr_.clear();
    attr_.emplace_back(PluginField("epsilon", nullptr, PluginFieldType::kFLOAT32, 1));
    fc_.nbFields = attr_.size();
    fc_.fields   = attr_.data();
}

LayerNormPluginV5Creator::~LayerNormPluginV5Creator()
{
    WHERE_AM_I();
}

IPluginV2 *LayerNormPluginV5Creator::createPlugin(const char *name, const PluginFieldCollection *fc) noexcept
{
    WHERE_AM_I();
    float                          epsilon = 1.0e-5f;
    std::map<std::string, float *> parameterMap {{"epsilon", &epsilon}};

    for (int i = 0; i < fc->nbFields; ++i)
    {
        if (parameterMap.find(fc->fields[i].name) != parameterMap.end())
        {
            *parameterMap[fc->fields[i].name] = *reinterpret_cast<const float *>(fc->fields[i].data);
        }
    }
    return new LayerNormPluginV5(name, epsilon);
}

IPluginV2 *LayerNormPluginV5Creator::deserializePlugin(const char *name, const void *serialData, size_t serialLength) noexcept
{
    WHERE_AM_I();
    return new LayerNormPluginV5(name, serialData, serialLength);
}

void LayerNormPluginV5Creator::setPluginNamespace(const char *pluginNamespace) noexcept
{
    WHERE_AM_I();
    namespace_ = pluginNamespace;
    return;
}

const char *LayerNormPluginV5Creator::getPluginNamespace() const noexcept
{
    WHERE_AM_I();
    return namespace_.c_str();
}

const char *LayerNormPluginV5Creator::getPluginName() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_NAME;
}

const char *LayerNormPluginV5Creator::getPluginVersion() const noexcept
{
    WHERE_AM_I();
    return PLUGIN_VERSION;
}

const PluginFieldCollection *LayerNormPluginV5Creator::getFieldNames() noexcept
{
    WHERE_AM_I();
    return &fc_;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginV5Creator);

} // namespace nvinfer1
