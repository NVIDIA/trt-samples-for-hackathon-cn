#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "CumSumPlugin.h"

using namespace nvinfer1;
using namespace plugin;

PluginFieldCollection    CumSumPluginCreator::mFC {};
std::vector<PluginField> CumSumPluginCreator::mPluginAttributes;

template<typename T>
__global__ void scanLastWarp(const T *input, T *output, int nWidth)
{
    const int bx = blockIdx.x, tx = threadIdx.x;
    //extern __shared__ T list[]; // compile error, need some trick
    extern __shared__ __align__(sizeof(T)) unsigned char byte[];
    T *                                                  list = reinterpret_cast<T *>(byte);
    if (tx >= nWidth)
        return;

    list[tx] = input[bx * nWidth + tx];
    typedef hipcub::WarpScan<T, 32>              WarpScan;
    __shared__ typename WarpScan::TempStorage tempScan;
    T &                                       tDataScan = list[tx];
    WarpScan(tempScan).InclusiveSum(tDataScan, tDataScan);

    output[bx * nWidth + tx] = list[tx];
}

template<typename T>
__global__ void scanLastBlock(const T *input, T *output, int nWidth)
{
    const int bx = blockIdx.x, tx = threadIdx.x;

    //extern __shared__ T row[]; // compile error, need some trick
    extern __shared__ __align__(sizeof(T)) unsigned char byte[];
    T *                                                  list = reinterpret_cast<T *>(byte);
    if (tx >= nWidth)
        return;

    list[tx] = input[bx * nWidth + tx];
    typedef hipcub::BlockScan<T, 1024>            BlockScan;
    __shared__ typename BlockScan::TempStorage tempScan;
    T &                                        tDataScan = list[tx];
    BlockScan(tempScan).InclusiveSum(tDataScan, tDataScan);
    __syncthreads();

    output[bx * nWidth + tx] = list[tx];
}

template<typename T>
__global__ void scanOther(const T *input, T *output, const int nLoop, const int nWidth)
{
    const int tx    = threadIdx.x;
    const int index = blockIdx.y * gridDim.x * nLoop * nWidth + blockIdx.x * nWidth + tx;

    if (tx >= nWidth)
        return;

    T sum = T(0);
    for (int i = 0; i < nLoop * gridDim.x * nWidth; i += gridDim.x * nWidth)
    {
        sum += input[index + i];
        output[index + i] = sum;
    }
}

int CumSumPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream)
{
    const int condition = int(m.nDim - m.axis == 1) * (1 + int(m.nWidth > 32)) * 4 + m.datatype;
#if DEBUG
    printf("nDim=%d,axis=%d,datatype=%d,nHighDim=%d,nLowDim=%d,nLoop=%d,nWidth=%d,kernelKind=%d,condition=%d\n", m.nDim, m.axis, m.datatype, m.nHighDim, m.nLowDim, m.nLoop, m.nWidth, m.kernelKind, condition);
#endif
    switch (condition)
    {
    case 0: // higher axis, float32
        (scanOther<float>)<<<dim3(m.nLowDim, m.nHighDim), ALIGN32(m.nWidth), 0, stream>>>((float *)inputs[0], (float *)outputs[0], m.nLoop, m.nWidth);
        break;
    case 1: // higher axis, float16
        (scanOther<__half>)<<<dim3(m.nLowDim, m.nHighDim), ALIGN32(m.nWidth), 0, stream>>>((__half *)inputs[0], (__half *)outputs[0], m.nLoop, m.nWidth);
        break;
    //case 2:   // higher axis, int8
    case 3: // higher axis, int32
        (scanOther<int>)<<<dim3(m.nLowDim, m.nHighDim), ALIGN32(m.nWidth), 0, stream>>>((int *)inputs[0], (int *)outputs[0], m.nLoop, m.nWidth);
        break;
    case 4: // last axis, width <= 32, float32
        (scanLastWarp<float>)<<<m.nHighDim, 32, sizeof(float) * 32, stream>>>((float *)inputs[0], (float *)outputs[0], m.nWidth);
        break;
    case 5: // last axis, width <= 32, float16
        (scanLastWarp<__half>)<<<m.nHighDim, 32, sizeof(__half) * 32, stream>>>((__half *)inputs[0], (__half *)outputs[0], m.nWidth);
        break;
    //case 6:   // last axis, width <= 32, int8
    case 7: // last axis, width <= 32, int32
        (scanLastWarp<int>)<<<m.nHighDim, 32, sizeof(int) * 32, stream>>>((int *)inputs[0], (int *)outputs[0], m.nWidth);
        break;
    case 8: // last axis, width > 32, float32
        (scanLastBlock<float>)<<<m.nHighDim, 1024, sizeof(float) * 1024, stream>>>((float *)inputs[0], (float *)outputs[0], m.nWidth);
        break;
    case 9: // last axis, width > 32, float16
        (scanLastBlock<__half>)<<<m.nHighDim, 1024, sizeof(__half) * 1024, stream>>>((__half *)inputs[0], (__half *)outputs[0], m.nWidth);
        break; // large kernel, float16
    //case 10:  // last axis, width > 32, int8
    case 11: // last axis, width > 32, int32
        (scanLastBlock<int>)<<<m.nHighDim, 1024, sizeof(int) * 1024, stream>>>((int *)inputs[0], (int *)outputs[0], m.nWidth);
        break; // large kernel, int32
    default:
#if DEBUG
        printf("[CumSumPlugin::enqueue()]Error condition! %d\n", condition);
#endif
        break;
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(CumSumPluginCreator);
