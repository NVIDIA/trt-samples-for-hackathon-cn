#include "hip/hip_runtime.h"
#include "LayerNormPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    LayerNormPluginCreator::fc_ {};
std::vector<PluginField> LayerNormPluginCreator::attr_;

__global__ void layerNormKernel(float *pInput, float *pOutput)
{
    const int tx = threadIdx.x, index = blockIdx.x * 256 + threadIdx.x;

    __shared__ float temp[128];

    float value0 = pInput[index];
    float value1 = pInput[index + 128];

    temp[tx] = value0 + value1;
    __syncthreads();

    for (int stride = 64; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            temp[tx] += temp[tx + stride];
        }
        __syncthreads();
    }
    float mean = temp[0] / 256;
    __syncthreads();

    temp[tx] = (value0 - mean) * (value0 - mean) + (value1 - mean) * (value1 - mean);
    __syncthreads();

    for (int stride = 64; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            temp[tx] += temp[tx + stride];
        }
        __syncthreads();
    }
    float var = temp[0] / 256;

    pOutput[index]       = (value0 - mean) * rsqrtf(var + EPSILON);
    pOutput[index + 128] = (value1 - mean) * rsqrtf(var + EPSILON);
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    const int nBlock = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1]; // 仅用于处理 nHiddenDimension 为 256 的情况

    layerNormKernel<<<nBlock, 128, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);
    return 0;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);
