#include "hip/hip_runtime.h"
#include "utils.h"
#define LOAD_FROM_FLOAT

using namespace std;
namespace std
{
template<typename _CharT, typename _Traits>
inline basic_ostream<_CharT, _Traits> &
tab(basic_ostream<_CharT, _Traits> &__os)
{
    return __os.put(__os.widen('\t'));
}
} // namespace std

std::string stringPadding(std::string original, size_t charCount)
{
    original.resize(charCount, ' ');
    return original;
}

/*************Error Handling**************/
bool check(hipError_t e, int iLine, const char *szFile)
{
    if (e != hipSuccess)
    {
        cout << "CUDA runtime API error " << hipGetErrorName(e) << " with e= " << e << " at line " << iLine << " in file " << szFile << endl;
        exit(0);
        return false;
    }
    return true;
}
const char *cublasGetErrorString(hipblasStatus_t status)
{
    switch (status)
    {
    case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN";
    }
    return "unknown error";
}

bool check(hipblasStatus_t e, int iLine, const char *szFile)
{
    if (e != HIPBLAS_STATUS_SUCCESS)
    {
        cout << "CUDA CUBLAS runtime API error " << cublasGetErrorString(e) << " with e= " << e << " at line " << iLine << " in file " << szFile << endl;
        exit(0);
        return false;
    }
    return true;
}

/*************Time Handling**************/
CudaTimer::CudaTimer(hipStream_t stream)
{
    this->stream = stream;
}

void CudaTimer::start()
{
#ifdef CHECK
    ck(hipEventCreate(&event_start));
    ck(hipEventCreate(&event_stop));
    ck(hipEventRecord(event_start, stream));
#else
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    hipEventRecord(event_start, stream);
#endif
}
float CudaTimer::stop()
{
#ifdef CHECK
    ck(hipEventRecord(event_stop, stream));
    ck(hipEventSynchronize(event_stop));
    ck(hipEventElapsedTime(&time, event_start, event_stop));
    ck(hipEventDestroy(event_start));
    ck(hipEventDestroy(event_stop));
#else
    hipEventRecord(event_stop, stream);
    hipEventSynchronize(event_stop);
    hipEventElapsedTime(&time, event_start, event_stop);
    hipEventDestroy(event_start);
    hipEventDestroy(event_stop);
#endif
    return time;
}
CudaTimer::~CudaTimer()
{
}

/*************Useful functions***********************/
int blockNum(int size, int blockSize)
{
    int nblock = (size - 1) / blockSize + 1;
    return nblock;
}
int next_pow2(int a)
{
    int rval = 32;
    if (a > 32)
    {
        while (rval < a)
            rval <<= 1;
    }
    return rval;
}

template<typename T>
int numPerThread()
{
    return sizeof(float) / sizeof(T);
}

template<typename T>
void deviceMalloc(T **ptr, int size)
{
    ck(hipMalloc((void **)ptr, sizeof(T) * size));
}

template<typename T>
void deviceMemset(T *ptr, int value, int size)
{
    ck(hipMemset((void *)ptr, 0, sizeof(T) * size));
}

template<typename T>
hipError_t deviceFree(T *&ptr)
{
    hipError_t res = hipSuccess;
    if (ptr != NULL)
    {
        res = hipFree(ptr);
        ptr = NULL;
    }
    return res;
}

template<typename T>
void deviceMemcpyHtoD(hipStream_t stream, T *d_ptr, T *h_ptr, int size)
{
    ck(hipMemcpyAsync(d_ptr, h_ptr, size * sizeof(T), hipMemcpyHostToDevice, stream));
}

template<typename T>
float castToFloat(T input)
{
    float output = (T)(input);
    return output;
}

template<>
float castToFloat(__half input)
{
    float output = __half2float(input);
    return output;
}

template<typename T>
void setRandom(T *data, int len)
{
    std::random_device                    rd;
    std::mt19937                          mt(rd());
    std::uniform_real_distribution<float> dist(1.0, 10.0);
    for (int i = 0; i < len; i++)
    {
        if (sizeof(T) == 4)
        {
            data[i] = static_cast<T>(dist(mt));
        }
        else
        {
            data[i] = __float2half_rn(static_cast<float>(dist(mt)));
        }
    }
}
/*********************Npz &Npy File Process functions***********************/

/*********************The explicit instantiation part***********************/
template int numPerThread<float>();
template int numPerThread<__half>();

template float castToFloat<float>(float input);
template float castToFloat<__half>(__half input);

template void        deviceMalloc<float>(float **ptr, int size);
template void        deviceMemset<float>(float *ptr, int value, int size);
template hipError_t deviceFree<float>(float *&ptr);
template void        deviceMemcpyHtoD<float>(hipStream_t stream, float *d_ptr, float *h_ptr, int size);

template void        deviceMalloc<int>(int **ptr, int size);
template void        deviceMemset<int>(int *ptr, int value, int size);
template hipError_t deviceFree<int>(int *&ptr);
template void        deviceMemcpyHtoD<int>(hipStream_t stream, int *d_ptr, int *h_ptr, int size);

template void        deviceMalloc<__half>(__half **ptr, int size);
template void        deviceMemset<__half>(__half *ptr, int value, int size);
template hipError_t deviceFree<__half>(__half *&ptr);
template void        deviceMemcpyHtoD<__half>(hipStream_t stream, __half *d_ptr, __half *h_ptr, int size);

template void        deviceMalloc<bool>(bool **ptr, int size);
template void        deviceMemset<bool>(bool *ptr, int value, int size);
template void        deviceMemcpyHtoD<bool>(hipStream_t stream, bool *d_ptr, bool *h_ptr, int size);
template hipError_t deviceFree<bool>(bool *&ptr);

template void setRandom<__half>(__half *data, int len);
template void setRandom<float>(float *data, int len);
template void setRandom<int>(int *data, int len);
template void setRandom<bool>(bool *data, int len);
